#include "hip/hip_runtime.h"
//#include "../../cub-1.8.0/hipcub/hipcub.hpp"   // or equivalently <cub/device/device_histogram.cuh>
#include "hipcub/hipcub.hpp"
#include "helper.cu.h"

void randomInitNat(uint32_t* data, const uint32_t size) {
    for (int i = 0; i < size; ++i) {
        uint32_t r = rand();
        data[i] = r;
    }
}

void randomInitShp(uint32_t* data, const uint32_t M, const uint32_t N) {
    for (int i = 0; i < M; i++) { // Initialize all entries of shp to 1.
        data[i] = 1;
    }
    for (int j = 0; j < N-M; ++j) { // Add 1 to a random entry in shp.
        unsigned long int r = rand();
        data[r % M] += 1;
    }
}

void randomInitKs(uint32_t* h_ks, uint32_t* h_shp, const uint32_t M) {
    for (int i = 0; i < M; i++) { // Set entries of k to be between zero and 2/3 length of that array.
        unsigned long int r = rand();
        h_ks[i] = r % ((h_shp[i]/3)*2);
    }
}


__global__ void
extractKthElem(const uint32_t m, uint32_t* d_offsets, uint32_t* ks, uint32_t* As_sorted, uint32_t* res) {
    uint32_t gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < m) { // Assuming: 0 <= gid < m
        res[gid] = As_sorted[d_offsets[gid]+ks[gid]-1];
    }
}

// rankSearchBatch (ks: [m]i32) (shp: [m]i32) (II1: *[n]i32) (A: [n]f32) : [m]f32 = ...
double simpleBatchRankSearchK(uint32_t m, uint32_t n,
                              uint32_t* d_ks,
                              uint32_t* h_shp,
                              uint32_t* d_As,
                              uint32_t* d_res){
    // Declare, allocate, and initialize device-accessible pointers
    // for sorting data
    uint32_t  num_items = n;
    uint32_t  num_segments = m;
    uint32_t  *h_offsets = (uint32_t*) malloc(sizeof(uint32_t) * (m+1));
    // Setup offsets
    h_offsets[0] = 0;
    for (uint32_t i = 1; i < m+1; i++) {
        h_offsets[i] = h_offsets[i-1] + h_shp[i-1];
    }
    uint32_t* d_offsets;
    cudaSucceeded(hipMalloc((void**) &d_offsets, sizeof(uint32_t) * (m+1)));
    cudaSucceeded(hipMemcpy(d_offsets, h_offsets, sizeof(uint32_t) * (m+1), hipMemcpyHostToDevice));

    // Setup in/out arrays
    uint32_t* d_keys_in = d_As;
    uint32_t* d_keys_out;
    cudaSucceeded(hipMalloc((void**) &d_keys_out, n * sizeof(uint32_t)));

    // Setup for the extractKthElem kernel
    uint32_t block_size = min(m, 1024);
    uint32_t num_blocks = (m + block_size - 1) / block_size;

    // ACTUALLY DOING RANK SEARCH K
    // Determine temporary device storage requirements
    
    void     *d_temp_storage = nullptr;
    size_t   temp_storage_bytes = 0;
    {   // Sort prelude
        hipcub::DeviceSegmentedRadixSort::SortKeys(
            d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out,
            num_items, num_segments, d_offsets, d_offsets + 1);

        // Allocate temporary storage
        cudaSucceeded(hipMalloc(&d_temp_storage, temp_storage_bytes));
    }
    cudaCheckError();
    {   // One dry run 
        // Run sorting operation
        hipcub::DeviceSegmentedRadixSort::SortKeys(
            d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out,
            num_items, num_segments, d_offsets, d_offsets + 1);
        // Extract the kth elements
        extractKthElem<<<num_blocks, block_size>>>(m, d_offsets, d_ks, d_keys_out, d_res);
    }
    hipDeviceSynchronize();
    cudaCheckError();

    // Timing
    double elapsed;
    struct timeval t_start, t_end, t_diff;
    gettimeofday(&t_start, NULL);

    for(int k=0; k<GPU_RUNS; k++) {
        // Run sorting operation
        hipcub::DeviceSegmentedRadixSort::SortKeys(
            d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out,
            num_items, num_segments, d_offsets, d_offsets + 1);
        // Extract the kth elements
        extractKthElem<<<num_blocks, block_size>>>(m, d_offsets, d_ks, d_keys_out, d_res);
    }
    hipDeviceSynchronize();

    gettimeofday(&t_end, NULL);
    timeval_subtract(&t_diff, &t_end, &t_start);
    elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / ((double)GPU_RUNS);

    cudaCheckError();
    // Validation
    uint32_t* h_keys_out = (uint32_t*) malloc(sizeof(uint32_t) * n);
    uint32_t* h_ks = (uint32_t*) malloc(sizeof(uint32_t) * m);
    uint32_t* h_res = (uint32_t*) malloc(sizeof(uint32_t) * m);
    cudaSucceeded(hipMemcpy(h_keys_out, d_keys_out, sizeof(uint32_t) * n, hipMemcpyDeviceToHost));
    cudaSucceeded(hipMemcpy(h_ks, d_ks, sizeof(uint32_t) * m, hipMemcpyDeviceToHost));
    cudaSucceeded(hipMemcpy(h_res, d_res, sizeof(uint32_t) * m, hipMemcpyDeviceToHost));

    // VALIDATION THAT THE OUTPUT ARRAY IS ACTUALLY SORTED!
    for (uint32_t j = 0; j < m; j++) {
        for (uint32_t i = h_offsets[j]+1; i < h_offsets[j+1]; i++) {
            if (h_keys_out[i-1] > h_keys_out[i]) {
                printf("INVALID RESULT for i:%d, (A[i-1]=%d > A[i]=%d)\n", i, h_keys_out[i-1], h_keys_out[i]);
                return elapsed;
            }
        }
    }
    // VALIDATION THAT THE CORRECT ELEMENTS ARE EXTRACTED
    for (uint32_t j = 0; j < m; j++) {
        if (h_res[j] != h_keys_out[h_offsets[j]+h_ks[j]-1]) {
            printf("INVALID RESULT for j:%d, (res[j]=%d != actual[j]=%d)\n", j, h_res[j], h_keys_out[h_offsets[j]+h_ks[j]-1]);
            return elapsed;
        }
    }
    printf("!!!VALID RESULT!!!\n");

    hipFree(d_temp_storage);
    hipFree(d_keys_out);
    hipFree(d_offsets);
    free(h_keys_out);
    free(h_offsets);
    free(h_res);
    free(h_ks);

    return elapsed;
}


int main (int argc, char * argv[]) {
    if (argc != 3) {
        printf("Usage: %s <size-of-flat-array> <size-of-shp>\n", argv[0]);
        exit(1);
    }
    const uint64_t N = atoi(argv[1]);
    const uint64_t M = atoi(argv[2]);

    //Allocate and Initialize Host data with random values
    uint32_t* h_keys  = (uint32_t*) malloc(N*sizeof(uint32_t));
    randomInitNat(h_keys, N);
    uint32_t* h_shp   = (uint32_t*) malloc(M*sizeof(uint32_t));
    randomInitShp(h_shp, M, N);
    uint32_t* h_ks   = (uint32_t*) malloc(M*sizeof(uint32_t));
    randomInitKs(h_ks, h_shp, M);

    //Allocate and Initialize Device data
    uint32_t* d_keys_in;
    uint32_t* d_ks;
    uint32_t* d_res;
    // uint32_t* d_keys_out;
    cudaSucceeded(hipMalloc((void**) &d_keys_in,  N * sizeof(uint32_t)));
    cudaSucceeded(hipMemcpy(d_keys_in, h_keys, N * sizeof(uint32_t), hipMemcpyHostToDevice));
    cudaSucceeded(hipMalloc((void**) &d_ks,  M * sizeof(uint32_t)));
    cudaSucceeded(hipMemcpy(d_ks, h_ks, M * sizeof(uint32_t), hipMemcpyHostToDevice));
    cudaSucceeded(hipMalloc((void**) &d_res,  M * sizeof(uint32_t)));
    
    printf("Batch Rank Search K for N=%lu and M=%lu\n", N, M);
    
    double elapsed = simpleBatchRankSearchK(M, N, d_ks, h_shp, d_keys_in, d_res);
    printf("Runs in: %.2f us\n", elapsed);

    //Cleanup and closing
    free(h_keys);
    free(h_shp);
    free(h_ks);
    hipFree(d_keys_in);
    hipFree(d_ks);
    hipFree(d_res);
    
    return 0;
}
